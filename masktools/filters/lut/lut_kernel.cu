#include "hip/hip_runtime.h"
#include <stdint.h>
#include <avisynth.h>
#include "CommonFunctions.h"
#include "VectorFunctions.cuh"

#ifndef NDEBUG
//#if 1
#define DEBUG_SYNC \
		CUDA_CHECK(hipGetLastError()); \
      CUDA_CHECK(hipDeviceSynchronize())
#else
#define DEBUG_SYNC
#endif

template<typename vpixel_t, typename pixel_t, int bits_per_pixel>
__global__ void kl_lut_x(vpixel_t* dst, const vpixel_t* __restrict__ src,
   int pitch4, int width4, int height, const pixel_t* __restrict__ lut, int mask)
{
   int x = threadIdx.x + blockIdx.x * blockDim.x;
   int y = threadIdx.y + blockIdx.y * blockDim.y;

   if (x < width4 && y < height) {
      auto X = src[x + y * pitch4];
      if (sizeof(pixel_t) == 1) {
         vpixel_t t = {
            lut[X.x],
            lut[X.y],
            lut[X.z],
            lut[X.w]
         };
         dst[x + y * pitch4] = t;
      }
      else { // == 2
         vpixel_t t = {
            lut[X.x & mask],
            lut[X.y & mask],
            lut[X.z & mask],
            lut[X.w & mask]
         };
         dst[x + y * pitch4] = t;
      }
   }
}

template<int bits_per_pixel>
__device__ int lut_index_xy(int x, int y) {
   return (x << bits_per_pixel) + y;
}

template<typename vpixel_t, typename pixel_t, int bits_per_pixel>
__global__ void kl_lut_xy(vpixel_t* dst,
   const vpixel_t* __restrict__ src0,
   const vpixel_t* __restrict__ src1,
   int pitch4, int width4, int height, const pixel_t* __restrict__ lut, int mask)
{
   int x = threadIdx.x + blockIdx.x * blockDim.x;
   int y = threadIdx.y + blockIdx.y * blockDim.y;

   if (x < width4 && y < height) {
      auto X = src0[x + y * pitch4];
      auto Y = src1[x + y * pitch4];
      if (sizeof(pixel_t) == 1) {
         vpixel_t t = {
            lut[lut_index_xy<bits_per_pixel>(X.x, Y.x)],
            lut[lut_index_xy<bits_per_pixel>(X.y, Y.y)],
            lut[lut_index_xy<bits_per_pixel>(X.z, Y.z)],
            lut[lut_index_xy<bits_per_pixel>(X.w, Y.w)]
         };
         dst[x + y * pitch4] = t;
      }
      else { // == 2
         vpixel_t t = {
            lut[lut_index_xy<bits_per_pixel>(X.x, Y.x) & mask],
            lut[lut_index_xy<bits_per_pixel>(X.y, Y.y) & mask],
            lut[lut_index_xy<bits_per_pixel>(X.z, Y.z) & mask],
            lut[lut_index_xy<bits_per_pixel>(X.w, Y.w) & mask]
         };
         dst[x + y * pitch4] = t;
      }
   }
}

template<int bits_per_pixel>
__device__ int lut_index_xyz(int x, int y, int z) {
   return (x << (bits_per_pixel * 2)) + (y << bits_per_pixel) + z;
}

template<typename vpixel_t, typename pixel_t, int bits_per_pixel>
__global__ void kl_lut_xyz(vpixel_t* dst,
   const vpixel_t* __restrict__ src0,
   const vpixel_t* __restrict__ src1,
   const vpixel_t* __restrict__ src2,
   int pitch4, int width4, int height, const pixel_t* __restrict__ lut, int mask)
{
   int x = threadIdx.x + blockIdx.x * blockDim.x;
   int y = threadIdx.y + blockIdx.y * blockDim.y;

   if (x < width4 && y < height) {
      auto X = src0[x + y * pitch4];
      auto Y = src1[x + y * pitch4];
      auto Z = src2[x + y * pitch4];
      if (sizeof(pixel_t) == 1) {
         vpixel_t t = {
            lut[lut_index_xyz<bits_per_pixel>(X.x, Y.x, Z.x)],
            lut[lut_index_xyz<bits_per_pixel>(X.y, Y.y, Z.y)],
            lut[lut_index_xyz<bits_per_pixel>(X.z, Y.z, Z.z)],
            lut[lut_index_xyz<bits_per_pixel>(X.w, Y.w, Z.w)]
         };
         dst[x + y * pitch4] = t;
      }
      else { // == 2
         vpixel_t t = {
            lut[lut_index_xyz<bits_per_pixel>(X.x, Y.x, Z.x) & mask],
            lut[lut_index_xyz<bits_per_pixel>(X.y, Y.y, Z.y) & mask],
            lut[lut_index_xyz<bits_per_pixel>(X.z, Y.z, Z.z) & mask],
            lut[lut_index_xyz<bits_per_pixel>(X.w, Y.w, Z.w) & mask]
         };
         dst[x + y * pitch4] = t;
      }
   }
}

template <typename vpixel_t, typename pixel_t, int bits_per_pixel>
void lut_cuda(int num_input, pixel_t *pDst, const pixel_t * const *pSrc, int pitch, int width, int height, const pixel_t* lut, PNeoEnv env)
{
   const int mask = (1 << bits_per_pixel) - 1;

   int width4 = width >> 2;
   int pitch4 = pitch >> 2;

   dim3 threads(16, 8);
   dim3 blocks(nblocks(width4, threads.x), nblocks(height, threads.y));

   switch (num_input) {
   case 1:
      kl_lut_x<vpixel_t, pixel_t, bits_per_pixel> << <blocks, threads >> > ((vpixel_t*)pDst,
         (const vpixel_t*)pSrc[0],
         pitch4, width4, height, lut, mask);
      DEBUG_SYNC;
      break;
   case 2:
      kl_lut_xy<vpixel_t, pixel_t, bits_per_pixel> << <blocks, threads >> > ((vpixel_t*)pDst,
         (const vpixel_t*)pSrc[0], (const vpixel_t*)pSrc[1],
         pitch4, width4, height, lut, mask);
      DEBUG_SYNC;
      break;
   case 3:
      kl_lut_xyz<vpixel_t, pixel_t, bits_per_pixel> << <blocks, threads >> > ((vpixel_t*)pDst,
         (const vpixel_t*)pSrc[0], (const vpixel_t*)pSrc[1], (const vpixel_t*)pSrc[2],
         pitch4, width4, height, lut, mask);
      DEBUG_SYNC;
      break;
   }
}

void lut_cuda_16(int bits_per_pixel, int num_input, uint16_t *pDst, const uint16_t * const *pSrc, int pitch, int width, int height, const uint16_t* lut, PNeoEnv env)
{
   switch (bits_per_pixel) {
   case 10: return lut_cuda<ushort4, uint16_t, 8>(num_input, pDst, pSrc, pitch, width, height, lut, env);
   case 12: return lut_cuda<ushort4, uint16_t, 8>(num_input, pDst, pSrc, pitch, width, height, lut, env);
   case 14: return lut_cuda<ushort4, uint16_t, 8>(num_input, pDst, pSrc, pitch, width, height, lut, env);
   case 16: return lut_cuda<ushort4, uint16_t, 8>(num_input, pDst, pSrc, pitch, width, height, lut, env);
   }
}

void lut_cuda(int bits_per_pixel, int num_input, uint8_t *pDst, const uint8_t * const *pSrc, int pitch, int width, int height, const void* lut, PNeoEnv env)
{
   if (bits_per_pixel == 8) {
      return lut_cuda<uchar4, uint8_t, 8>(num_input, pDst, pSrc, pitch, width, height, (const uint8_t*)lut, env);
   }
   else {
      return lut_cuda_16(bits_per_pixel, num_input,
         (uint16_t*)pDst, (const uint16_t * const *)pSrc, pitch, width, height, (const uint16_t*)lut, env);
   }
}
