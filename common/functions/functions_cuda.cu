#include "hip/hip_runtime.h"
#include "functions.h"
#include "CommonFunctions.h"
#include "VectorFunctions.cuh"

#ifndef NDEBUG
//#if 1
#define DEBUG_SYNC \
		CUDA_CHECK(hipGetLastError()); \
      CUDA_CHECK(hipDeviceSynchronize())
#else
#define DEBUG_SYNC
#endif

using namespace Filtering;

template <typename vpixel_t, typename pixel_t>
__global__ void kl_fill(vpixel_t* dst, pixel_t v, int width, int height, int pitch)
{
   int x = threadIdx.x + blockIdx.x * blockDim.x;
   int y = threadIdx.y + blockIdx.y * blockDim.y;

   if (x < width && y < height) {
      dst[x + y * pitch] = VHelper<vpixel_t>::make(v);
   }
}

void Functions::memset_plane_cuda(Byte *ptr, ptrdiff_t pitch, int width, int height, Byte value, IScriptEnvironment* env)
{
   UNUSED(env);
   int p4 = (int)pitch >> 2;
   int w4 = (int)width >> 2;
   dim3 threads(32, 16);
   dim3 blocks(nblocks(w4, threads.x), nblocks(height, threads.y));
   kl_fill<<<blocks, threads>>>((uchar4*)ptr, value, w4, height, p4);
   DEBUG_SYNC;
}

void Functions::memset_plane_16_cuda(Byte *ptr, ptrdiff_t pitch, int width, int height, Word value, IScriptEnvironment* env)
{
   UNUSED(env);
   int p4 = (int)pitch >> 3;
   int w4 = (int)width >> 2;
   dim3 threads(32, 16);
   dim3 blocks(nblocks(w4, threads.x), nblocks(height, threads.y));
   kl_fill << <blocks, threads >> >((ushort4*)ptr, value, w4, height, p4);
   DEBUG_SYNC;
}

void Functions::memset_plane_32_cuda(Byte *ptr, ptrdiff_t pitch, int width, int height, float value, IScriptEnvironment* env)
{
   UNUSED(env);
   int p4 = (int)pitch >> 4;
   int w4 = (int)width >> 2;
   dim3 threads(32, 16);
   dim3 blocks(nblocks(w4, threads.x), nblocks(height, threads.y));
   kl_fill << <blocks, threads >> >((float4*)ptr, value, w4, height, p4);
   DEBUG_SYNC;
}

template <typename pixel_t>
__global__ void kl_copy(
   pixel_t* dst, int dst_pitch, const pixel_t* __restrict__ src, int src_pitch, int width, int height)
{
   int x = threadIdx.x + blockIdx.x * blockDim.x;
   int y = threadIdx.y + blockIdx.y * blockDim.y;

   if (x < width && y < height) {
      dst[x + y * dst_pitch] = src[x + y * src_pitch];
   }
}

void Functions::copy_plane_cuda(Byte *pDst, ptrdiff_t dst_pitch, const Byte *pSrc, ptrdiff_t src_pitch, int rowsize, int height, IScriptEnvironment* env)
{
   UNUSED(env);
   int sp4 = (int)src_pitch >> 2;
   int dp4 = (int)dst_pitch >> 2;
   int w4 = rowsize >> 2;
   dim3 threads(32, 16);
   dim3 blocks(nblocks(w4, threads.x), nblocks(height, threads.y));
   kl_copy << <blocks, threads >> >((uchar4*)pDst, dp4, (uchar4*)pSrc, sp4, w4, height);
   DEBUG_SYNC;
}
